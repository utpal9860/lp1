#include<stdio.h>
#include<hip/hip_runtime.h>
__global__ void add(int *a,int *b,int *c)
{
	  int id = blockIdx.x*blockDim.x+threadIdx.x;
    if(id<5)
    	c[id] = a[id] + b[id];
}

 int main()
{
    const int arraySize = 5;
    float avg,sd;
	int d[arraySize];
 
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };
int *d_a,*d_b,*d_c,*d_d,i;
int size=sizeof(int)*arraySize;
hipMalloc((void **)&d_a,size);
hipMalloc((void **)&d_b,size);
hipMalloc((void **)&d_c,size);
hipMalloc((void **)&d_d,size);
int blocksize=1024;
int threadsize=(int) ceil ((float)arraySize/blocksize);
hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);
add<<<blocksize,threadsize>>>(d_a,d_b,d_c);
hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);

for(i=0;i<arraySize;i++)
{
printf("%d\t",c[i]);
}
int sum=0;
for(i=0;i<arraySize;i++)
{
sum=sum+c[i];
}
printf("Sum=%d\n",sum);
avg=sum/arraySize;
printf("Avg=%f\n",avg);
for(int i=0;i<arraySize;i++)
{
 d[i]=c[i]-avg;
 d[i]=d[i]*d[i];
sum=sum+d[i];
}
sum=sum/arraySize;
sd=sqrt(sum);
printf("sd=%f\n",sd);
hipFree(d_a);
hipFree(d_b);
hipFree(d_c);
hipFree(d_d);
return 0;
}
/*OUTPUT
cpllab00@cpllab:~$ nvcc stdev.cu
cpllab00@cpllab:~$ ./a.out
11	22	33	44	55	Sum=165
Avg=33.000000
sd=16.583124
*/

