#include<stdio.h>
#include<hip/hip_runtime.h>
# define s 1000

__global__ void min(int *a,int *c)
{
int id=threadIdx.x;
   *c=a[0];
  if(a[id]<*c)
    {
      *c=a[id];
    }
}

__global__ void max(int *a,int *d)
{
int id=threadIdx.x;
   *d=a[0];
  if(a[id]>*d)
    {
      *d=a[id];
    }
}

int main()
{
int i,a[s],c,d;
int *dev_a,*dev_c,*dev_d;
hipMalloc((void **) &dev_a, s*sizeof(int));			
hipMalloc((void **) &dev_c, s*sizeof(int));
hipMalloc((void **) &dev_d, s*sizeof(int));

for(i=0;i<s;i++)
{
a[i]=rand()%1000+1;

}
hipMemcpy(dev_a,a,s*sizeof(int),hipMemcpyHostToDevice);
 
min<<<1,s>>>(dev_a,dev_c);
max<<<1,s>>>(dev_a,dev_d);

hipMemcpy(&c, dev_c, s*sizeof(int),hipMemcpyDeviceToHost);
hipMemcpy(&d, dev_d, s*sizeof(int),hipMemcpyDeviceToHost);

printf("min=%d",c);
printf("max=%d",d);
hipFree(dev_a);
hipFree(dev_c);
hipFree(dev_d);
printf(" ");

return 0;
}

/*
OUTPUT
cpllab00@cpllab:~$ nvcc minmax.cu
cpllab00@cpllab:~$ ./a.out
min=251 max=578 */
